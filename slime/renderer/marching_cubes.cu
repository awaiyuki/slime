#include "hip/hip_runtime.h"
#include "marching_cubes.cuh"
#include "marching_cubes_tables.h"
#include <slime/constants/marching_cubes_constants.h>
#include <slime/utility/cuda_debug.cuh>
#include <bitset>
#include <iostream>

using namespace slime;
using namespace slime::MarchingCubesConstants;
using namespace std;

__device__ int *d_counter;

MarchingCubes::MarchingCubes(int _gridSize) : gridSize(_gridSize) {

  vertexData.vertices = new float3[gridSize * gridSize * gridSize * 15];
  vertexData.size = 0;

  float3 *d_vertices;
  hipMalloc((void **)&d_vertices,
             sizeof(float3) * gridSize * gridSize * gridSize * 15);

  VertexData tempVertexData;
  tempVertexData.size = 0;
  tempVertexData.vertices = d_vertices;
  hipMalloc((void **)&d_vertexDataPtr, sizeof(VertexData));
  hipMemcpy(d_vertexDataPtr, &tempVertexData, sizeof(VertexData),
             hipMemcpyHostToDevice);

  /* copy constant arrays */
  hipMemcpyToSymbol(HIP_SYMBOL(d_triangulation), MarchingCubesTables::triangulation,
                     sizeof(int) * 256 * 16);
  hipMemcpyToSymbol(HIP_SYMBOL(d_cornerIndexFromEdge),
                     MarchingCubesTables::cornerIndexFromEdge,
                     sizeof(int) * 12 * 2);

  hipMalloc((void **)&d_counter, sizeof(int));
}

MarchingCubes::~MarchingCubes() {
  if (vertexData.vertices) {
    delete[] vertexData.vertices;
  }
  if (d_vertexDataPtr) {
    hipFree(d_vertexDataPtr);
  }
  if (d_counter) {
    hipFree(d_counter);
  }
}

void MarchingCubes::march(hipGraphicsResource_t cudaVBOResource,
                          float *d_scalarField, float surfaceLevel) {

  hipMemset(d_counter, 0, sizeof(int));

  const int threadSize = THREAD_SIZE_IN_MARCH;
  dim3 dimBlock(threadSize, threadSize, threadSize);
  const int blockSize = (gridSize + threadSize - 1) / threadSize;
  dim3 dimGrid(blockSize, blockSize, blockSize);

  // hipMemset(d_vertexDataPtr->vertices, 0.0, sizeof(float3) * gridSize *
  // gridSize * gridSize * 15);
  g_march<<<dimGrid, dimBlock>>>(d_scalarField, gridSize, surfaceLevel,
                                 d_vertexDataPtr, d_counter);

  hipDeviceSynchronize();
  printCudaError("g_march");

  // Ensure d_counter is updated
  int h_counter;
  hipError_t err =
      hipMemcpy(&h_counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("hipMemcpy error: %s\n", hipGetErrorString(err));
  } else {
    printf("h_counter = %d\n", h_counter);
  }

  if (h_counter == 0) {
    return;
  }

  /* get triangles from device and return */
  // VertexData tempVertexData;
  // hipMemcpy(&tempVertexData, d_vertexDataPtr, sizeof(VertexData),
  //            hipMemcpyDeviceToHost);
  // hipMemcpy(vertexData.vertices, tempVertexData.vertices,
  //            sizeof(float3) * gridSize * gridSize * gridSize * 15,
  //            hipMemcpyDeviceToHost);
  // int h_counter;
  // hipMemcpy(&h_counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
  // // vertexData.size = gridSize * gridSize * gridSize * 15;
  // cout << h_counter << endl;
  // vertexData.size = h_counter;
  // //   hipMemcpy(&vertexData.size, d_counter, sizeof(int),
  // //   hipMemcpyDeviceToHost);
  // cout << vertexData.size << endl;
  // cout << vertexData.vertices[0].x << endl;

  /* CUDA-OpenGL interop */
  hipGraphicsMapResources(1, &cudaVBOResource, 0);
  float *d_positions;
  size_t size;
  hipGraphicsResourceGetMappedPointer((void **)&d_positions, &size,
                                       cudaVBOResource);
  printCudaError("hipGraphicsResourceGetMappedPointer in marching cubes");

  cout << "cudavboresource size: " << size << endl;

  int totalElements = gridSize * gridSize * gridSize * 15;
  int numThreads = THREAD_SIZE_IN_COPY_VERTEX_DATA;
  int numBlocks = (h_counter + numThreads - 1) / numThreads;

  g_copyVertexDataToVBO<<<numBlocks, numThreads>>>(d_positions, d_vertexDataPtr,
                                                   h_counter);
  hipDeviceSynchronize();
  printCudaError("copyVertexDataToVBODevice");

  hipGraphicsUnmapResources(1, &cudaVBOResource, 0);
};