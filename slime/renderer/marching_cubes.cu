#include "hip/hip_runtime.h"
#include "marching_cubes.cuh"
#include "marching_cubes_tables.h"

#include <bitset>
#include <iostream>

using namespace slime;
using namespace std;

__device__ int *d_counter;

MarchingCubes::MarchingCubes(int _gridSize) : gridSize(_gridSize) {

  vertexData.vertices = new float3[gridSize * gridSize * gridSize * 15];
  vertexData.size = 0;

  float3 *d_vertices;
  hipMalloc((void **)&d_vertices,
             sizeof(float3) * gridSize * gridSize * gridSize * 15);

  VertexData tempVertexData;
  tempVertexData.size = 0;
  tempVertexData.vertices = d_vertices;
  hipMalloc((void **)&d_vertexDataPtr, sizeof(VertexData));
  hipMemcpy(d_vertexDataPtr, &tempVertexData, sizeof(VertexData),
             hipMemcpyHostToDevice);

  /* copy constant arrays */
  hipMemcpyToSymbol(HIP_SYMBOL(d_triangulation), MarchingCubesTables::triangulation,
                     sizeof(int) * 256 * 16);
  hipMemcpyToSymbol(HIP_SYMBOL(d_cornerIndexFromEdge),
                     MarchingCubesTables::cornerIndexFromEdge,
                     sizeof(int) * 12 * 2);

  hipMalloc((void **)&d_counter, sizeof(int));
}

MarchingCubes::~MarchingCubes() {
  if (vertexData.vertices) {
    delete[] vertexData.vertices;
  }
  if (d_vertexDataPtr) {
    hipFree(d_vertexDataPtr);
  }
  if (d_counter) {
    hipFree(d_counter);
  }
}

VertexData MarchingCubes::march(float *d_scalarField, float surfaceLevel) {

  hipMemset(d_counter, 0, sizeof(int));

  const int threadSize = 8;
  dim3 dimBlock(threadSize, threadSize, threadSize);
  const int blockSize = (gridSize + threadSize - 1) / threadSize;
  dim3 dimGrid(blockSize, blockSize, blockSize);

  // hipMemset(d_vertexDataPtr->vertices, 0.0, sizeof(float3) * gridSize *
  // gridSize * gridSize * 15);
  marchParallel<<<dimGrid, dimBlock>>>(d_scalarField, gridSize, surfaceLevel,
                                       d_vertexDataPtr, d_counter);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("marchParallel error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();

  /* get triangles from device and return */
  VertexData tempVertexData;
  hipMemcpy(&tempVertexData, d_vertexDataPtr, sizeof(VertexData),
             hipMemcpyDeviceToHost);
  hipMemcpy(vertexData.vertices, tempVertexData.vertices,
             sizeof(float3) * gridSize * gridSize * gridSize * 15,
             hipMemcpyDeviceToHost);
  int h_counter;
  hipMemcpy(&h_counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
  // vertexData.size = gridSize * gridSize * gridSize * 15;
  cout << h_counter << endl;
  vertexData.size = h_counter;
  //   hipMemcpy(&vertexData.size, d_counter, sizeof(int),
  //   hipMemcpyDeviceToHost);
  cout << vertexData.size << endl;
  // cout << vertexData.vertices[0].x << endl;
  return vertexData;
};