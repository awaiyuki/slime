#include "hip/hip_runtime.h"
#include "marching_cubes.cuh"
#include "marching_cubes_tables.h"

#include <bitset>
#include <iostream>

using namespace slime;
using namespace std;

__device__ int* d_counter;

MarchingCubes::MarchingCubes(int _gridSize) : gridSize(_gridSize) {

  vertexData.vertices = new float3[gridSize * gridSize * gridSize*15];
  vertexData.size = 0;

  float3 *d_vertices;
  hipMalloc((void **)&d_vertices,
             sizeof(float3) * gridSize * gridSize * gridSize * 15);
  VertexData vertexData;
  vertexData.size = 0;
  vertexData.vertices = d_vertices;
  hipMalloc((void**)&d_vertexDataPtr, sizeof(VertexData));
  hipMemcpy(d_vertexDataPtr, &vertexData, sizeof(VertexData),
      hipMemcpyHostToDevice);
  
}

MarchingCubes::~MarchingCubes() {
  // hipFree(d_vertices);
}

VertexData MarchingCubes::march(float *d_scalarField, float surfaceLevel) {

  /* copy constant arrays */
  hipMemcpyToSymbol(HIP_SYMBOL(d_triangulation), MarchingCubesTables::triangulation,
                     sizeof(int) * 256 * 16);
  hipMemcpyToSymbol(HIP_SYMBOL(d_cornerIndexFromEdge),
                     MarchingCubesTables::cornerIndexFromEdge,
                     sizeof(int) * 12 * 2);

  hipMalloc((void**)&d_counter, sizeof(int));
  hipMemset(d_counter, 0, sizeof(int));

  const int threadSize = 8;
  dim3 dimBlock(threadSize, threadSize, threadSize);
  const int blockSize = (gridSize + threadSize - 1) / threadSize;
  dim3 dimGrid(blockSize, blockSize, blockSize);
  
  // hipMemset(d_vertexDataPtr->vertices, 0.0, sizeof(float3) * gridSize * gridSize * gridSize * 15);
  marchParallel<<<dimGrid, dimBlock>>>(d_scalarField, gridSize, surfaceLevel,
                                       d_vertexDataPtr, d_counter);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      printf("marchParallel error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();
  /* get triangles from device and return */
  VertexData tempVertexData;
  hipMemcpy(&tempVertexData, d_vertexDataPtr, sizeof(vertexData),
             hipMemcpyDeviceToHost);
  hipMemcpy(vertexData.vertices, tempVertexData.vertices,
             sizeof(float3) * gridSize * gridSize * gridSize * 15, hipMemcpyDeviceToHost);
  vertexData.size = gridSize * gridSize * gridSize * 15;

//  cout << vertexData.size << endl;
  //cout << vertexData.vertices[0].x << endl;
  return vertexData;
};