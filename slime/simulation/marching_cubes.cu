#include "marching_cubes.cuh"

#include <bitset>
#include <iostream>

using namespace slime;
using namespace std;

MarchingCubes::MarchingCubes(int gridSize) {
  hipMalloc((void **)&d_scalarField,
             sizeof(float) * gridSize * gridSize * gridSize);
}

MarchingCubes::~MarchingCubes() {}

std::vector<MarchingCubes::Triangle> MarchingCubes::march(float *scalarField,
                                                          float surfaceLevel) {

  /* triangles to scalarField, device */
  hipMemcpy(d_scalarField, scalarField,
             sizeof(float) * gridSize * gridSize * gridSize,
             hipMemcpyHostToDevice);
  hipMemcpy(...);
  /* copy constant arrays */

  const int threadSize = 128;
  dim3 dimBlock(threadSize, threadSize, threadSize);
  const int blockSize = (gridSize + threadSize - 1) / threadSize;
  dim3 dimGrid(blockSize, blockSize, blockSize);
  marchParallel<<<dimGrid, dimBlock>>>(d_scalarField, gridSize, surfaceLevel,
                                       d_triangles);
  hipDeviceSynchronize();
  /* get triangles from device and return */
  hipMemcpy(...);
};