#include "marching_cubes.cuh"

#include <bitset>
#include <iostream>

using namespace slime;
using namespace std;

MarchingCubes::MarchingCubes(int _gridSize) : gridSize(_gridSize) {

  vertexData.vertices = new glm::vec3[gridSize * gridSize * gridSize];
  vertexData.size = 0;

  glm::vec3 *d_vertices;
  hipMalloc((void **)&d_vertices,
             sizeof(glm::vec3) * gridSize * gridSize * gridSize);

  VertexData d_vertexData;
  d_vertexData.size = 0;
  d_vertexData.vertices = d_vertices;
  hipMalloc((void **)&d_vertexDataPtr, sizeof(VertexData));
  hipMemcpy(d_vertexDataPtr, &d_vertexData, sizeof(VertexData),
             hipMemcpyHostToDevice);
}

MarchingCubes::~MarchingCubes() {
  // hipFree(d_vertices);
}

VertexData MarchingCubes::march(float *d_scalarField, float surfaceLevel) {

  /* copy constant arrays */
  hipMemcpyToSymbol(HIP_SYMBOL(d_triangulation), MarchingCubesTables::triangulation,
                     sizeof(int) * 256 * 16);
  hipMemcpyToSymbol(HIP_SYMBOL(d_cornerIndexFromEdge),
                     MarchingCubesTables::cornerIndexFromEdge,
                     sizeof(int) * 12 * 2);

  const int threadSize = 128;
  dim3 dimBlock(threadSize, threadSize, threadSize);
  const int blockSize = (gridSize + threadSize - 1) / threadSize;
  dim3 dimGrid(blockSize, blockSize, blockSize);
  marchParallel<<<dimGrid, dimBlock>>>(d_scalarField, gridSize, surfaceLevel,
                                       d_vertexDataPtr);
  hipDeviceSynchronize();
  /* get triangles from device and return */
  VertexData tempVertexData;
  hipMemcpy(&tempVertexData, d_vertexDataPtr, sizeof(vertexData),
             hipMemcpyDeviceToHost);
  hipMemcpy(vertexData.vertices, tempVertexData.vertices,
             sizeof(glm::vec3) * tempVertexData.size, hipMemcpyDeviceToHost);
  vertexData.size = tempVertexData.size;

  cout << vertexData.size << endl;
  cout << vertexData.vertices[0].x;
  return vertexData;
};