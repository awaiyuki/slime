#include "hip/hip_runtime.h"
#include "marching_cubes_parallel.cuh"
#include "marching_cubes_tables.h"

using namespace slime;

__constant__ int slime::d_triangulation[256][16];
__constant__ int slime::d_cornerIndexFromEdge[12][2];

__device__ const int diff[8][3] = { { 0, 0, 0 }, { 1, 0, 0 }, { 1, 0, 1 },
    { 0, 0, 1 }, { 0, 1, 0 }, { 1, 1, 0 },
    { 1, 1, 1 }, { 0, 1, 1 }
};
__device__ float3 slime::interpolateVertices(float *d_scalarField, int gridSize,
                                             float surfaceLevel, int va[3],
                                             int vb[3]) {
  float scalarA =
      d_scalarField[va[2] * gridSize * gridSize + va[1] * gridSize + va[0]];
  float scalarB =
      d_scalarField[vb[2] * gridSize * gridSize + vb[1] * gridSize + vb[0]];
  float t = (surfaceLevel - scalarA) / (scalarB - scalarA);
  //printf("%d %d %f %f %f\n", va[0], vb[0], scalarA, scalarB, t);
  return make_float3(va[0], va[1], va[2]) +
         t * (make_float3(vb[0], vb[1], vb[2]) -
              make_float3(va[0], va[1], va[2]));
}

__global__ void slime::marchParallel(float *d_scalarField, int gridSize,
                                     float surfaceLevel,
                                     slime::VertexData *d_vertexDataPtr, int *d_counter) {

    

  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  int z = threadIdx.z + blockDim.z * blockIdx.z;
  printf("%f \n",
      d_scalarField[z * gridSize * gridSize + y * gridSize + x]);

  float3 currentPosition = make_float3(x, y, z);
  float3 cubeVertices[8];
  int cubeVertexCoordInt[8][3];

  for (int i = 0; i < 8; i++) {
    cubeVertices[i] =
        currentPosition + make_float3(diff[i][0], diff[i][1], diff[i][2]);
    cubeVertexCoordInt[i][0] = x + diff[i][0];
    cubeVertexCoordInt[i][1] = y + diff[i][1];
    cubeVertexCoordInt[i][2] = z + diff[i][2];
  }

  uint8_t tableKey = 0;
  for (int i = 0; i < 8; i++) {
    if (d_scalarField[(z + diff[i][2]) * gridSize * gridSize +
                      (y + diff[i][1]) * gridSize + (x + diff[i][0])] <
        surfaceLevel) { // correct?
      tableKey |= 1 << i;
    }
  }
  int* edges = d_triangulation[tableKey];

  for (int i = 0; i < 16; i += 3) {
    if (edges[i] == -1)
      continue;
    float3 v1Float3 = interpolateVertices(
        d_scalarField, gridSize, surfaceLevel,
        cubeVertexCoordInt[d_cornerIndexFromEdge[edges[i]][0]],
        cubeVertexCoordInt[d_cornerIndexFromEdge[edges[i]][1]]);

    glm::vec3 v1(v1Float3.x, v1Float3.y, v1Float3.z);

    float3 v2Float3 = interpolateVertices(
        d_scalarField, gridSize, surfaceLevel,
        cubeVertexCoordInt[d_cornerIndexFromEdge[edges[i + 1]][0]],
        cubeVertexCoordInt[d_cornerIndexFromEdge[edges[i + 1]][1]]);

    glm::vec3 v2(v2Float3.x, v2Float3.y, v2Float3.z);

    float3 v3Float3 = interpolateVertices(
        d_scalarField, gridSize, surfaceLevel,
        cubeVertexCoordInt[d_cornerIndexFromEdge[edges[i + 2]][0]],
        cubeVertexCoordInt[d_cornerIndexFromEdge[edges[i + 2]][1]]);

    glm::vec3 v3(v3Float3.x, v3Float3.y, v3Float3.z);

    // Need to correct!
    d_vertexDataPtr->vertices[*d_counter] = v1;
    d_vertexDataPtr->vertices[*d_counter + 1] = v2;
    d_vertexDataPtr->vertices[*d_counter + 2] = v3;

    atomicAdd(d_counter, 3);
    d_vertexDataPtr->size = *d_counter;
    // std::cout << "extract surface, triangle.v1[0]: " <<
    // triangle.v1[0]
    //           << std::endl;
  }
}