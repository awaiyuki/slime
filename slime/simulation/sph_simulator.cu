#include "hip/hip_runtime.h"

#include "sph_simulator.cuh"
#include "sph_simulator_device.cuh"
#include <cstring>
#include <random>
#include <iostream>
#include <stdio.h>
#include <glad/gl.h>
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>

using namespace slime;
using namespace std;

SPHSimulator::SPHSimulator(const unsigned int vbo) {
  random_device rd;
  mt19937 gen(rd());
  uniform_real_distribution<> dis(0.4f, 0.5f);

  for (int i = 0; i < SPHSimulatorConstants::NUM_PARTICLES; i++) {
    Particle particle;
    particle.id = i;

    float x = static_cast<float>(dis(gen));
    float y = static_cast<float>(dis(gen));
    float z = static_cast<float>(dis(gen));
    particle.position = make_float3(x, y, z);
    particle.velocity = make_float3(0, 0, 0);

    // cout << "initial position: " << x << y << z << endl;
    particle.mass = SPHSimulatorConstants::PARTICLE_MASS;
    particles.push_back(particle);
  }

  marchingCubes = make_unique<MarchingCubes>(GRID_SIZE);

  memset(scalarField, 0, sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE);

  hipMalloc((void **)&d_particles,
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES);
  hipMalloc((void **)&d_scalarField,
             sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE);

  hipMemcpy(d_particles, particles.data(),
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES,
             hipMemcpyHostToDevice);
  hipGraphicsGLRegisterBuffer(&cudaVBOResource, vbo, cudaGraphicsMapFlagsNone);
}

SPHSimulator::~SPHSimulator() {
  hipFree(d_particles);
  hipFree(d_scalarField);
}

std::vector<Particle> *SPHSimulator::getParticlesPointer() {
  return &particles;
}

void SPHSimulator::updateParticles(double deltaTime) {

  const int threadSize = 128;
  const int blockSize =
      (SPHSimulatorConstants::NUM_PARTICLES + threadSize - 1) / threadSize;

  computeDensityDevice<<<blockSize, threadSize>>>(d_particles);
  hipDeviceSynchronize();

  computePressureDevice<<<blockSize, threadSize>>>(d_particles);

  computePressureForceDevice<<<blockSize, threadSize>>>(d_particles, deltaTime);
  hipDeviceSynchronize();

  computeViscosityForceDevice<<<blockSize, threadSize>>>(d_particles,
                                                         deltaTime);
  hipDeviceSynchronize();

  computeSurfaceTensionDevice<<<blockSize, threadSize>>>(d_particles,
                                                         deltaTime);
  hipDeviceSynchronize();

  /*
  computeSurfaceTensionForce<<<blockSize, threadSize>>>(d_particles,
                                                         deltaTime);
  hipDeviceSynchronize();
  */

  computeGravityDevice<<<blockSize, threadSize>>>(d_particles, deltaTime);
  hipDeviceSynchronize();

  computePositionDevice<<<blockSize, threadSize>>>(d_particles, deltaTime);
  hipDeviceSynchronize();

  computeWallConstraintDevice<<<blockSize, threadSize>>>(d_particles,
                                                         deltaTime);
  hipDeviceSynchronize();

  updateSpatialHash<<<blockSize, threadSize>>>(d_particles);
  hipDeviceSynchronize();

  /* Copy Particle Positions to VBO positions array */
  hipGraphicsMapResources(1, &cudaVBOResource, 0);
  float *d_positions;
  size_t size;
  hipGraphicsResourceGetMappedPointer((void **)&d_positions, &size,
                                       cudaVBOResource);
  copyPositionToVBO<<<blockSize, threadSize>>>(d_positions, d_particles);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("cuda kernel error: %s\n", hipGetErrorString(err));
  }

  hipGraphicsUnmapResources(1, &cudaVBOResource, 0);
}

void SPHSimulator::updateScalarField() {
  /* Need to debug */

  const int threadSize = 8;
  dim3 dimBlock(threadSize, threadSize, threadSize);
  const int blockSize = (GRID_SIZE + threadSize - 1) / threadSize;
  dim3 dimGrid(blockSize, blockSize, blockSize);

  updateScalarFieldDevice<<<dimGrid, dimBlock>>>(d_scalarField, d_particles,
                                                 GRID_SIZE, 1954.0);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("updated_scalarField error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}

VertexData SPHSimulator::extractSurface() {
  return marchingCubes->march(d_scalarField,
                              SPHSimulatorConstants::SURFACE_LEVEL);
}
