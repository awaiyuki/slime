#include "hip/hip_runtime.h"

#include "sph_simulator.cuh"
#include "sph_simulator_device.cuh"
#include <cstring>
#include <random>
#include <iostream>
#include <stdio.h>
#include <glad/gl.h>
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>

using namespace slime;
using namespace std;

SPHSimulator::SPHSimulator(const unsigned int vbo) {
  random_device rd;
  mt19937 gen(rd());
  uniform_real_distribution<> dis(0.4f, 0.5f);

  for (int i = 0; i < SPHSimulatorConstants::NUM_PARTICLES; i++) {
    Particle particle;
    particle.id = i;

    float x = static_cast<float>(dis(gen));
    float y = static_cast<float>(dis(gen));
    float z = static_cast<float>(dis(gen));
    particle.position = make_float3(x, y, z);
    particle.velocity = make_float3(0, 0, 0);

    // cout << "initial position: " << x << y << z << endl;
    particle.mass = SPHSimulatorConstants::PARTICLE_MASS;
    particles.push_back(particle);
  }

  marchingCubes = make_unique<MarchingCubes>(GRID_SIZE);

  memset(colorField, 0, sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE);

  hipMalloc((void **)&particlesDevice,
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES);
  hipMalloc((void **)&colorFieldDevice,
             sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE);

  hipMemcpy(particlesDevice, particles.data(),
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES,
             hipMemcpyHostToDevice);
  hipGraphicsGLRegisterBuffer(&cudaVBOResource, vbo, cudaGraphicsMapFlagsNone);
}

SPHSimulator::~SPHSimulator() {
  hipFree(particlesDevice);
  hipFree(colorFieldDevice);
}

std::vector<Particle> *SPHSimulator::getParticlesPointer() {
  return &particles;
}

void SPHSimulator::updateParticles(double deltaTime) {

  const int threadSize = 128;
  const int blockSize =
      (SPHSimulatorConstants::NUM_PARTICLES + threadSize - 1) / threadSize;

  computeDensityDevice<<<blockSize, threadSize>>>(particlesDevice);
  hipDeviceSynchronize();

  computePressureDevice<<<blockSize, threadSize>>>(particlesDevice);

  computePressureForceDevice<<<blockSize, threadSize>>>(particlesDevice,
                                                        deltaTime);
  hipDeviceSynchronize();

  computeViscosityForceDevice<<<blockSize, threadSize>>>(particlesDevice,
                                                         deltaTime);
  hipDeviceSynchronize();

  computeSurfaceTensionDevice<<<blockSize, threadSize>>>(particlesDevice,
                                                         deltaTime);
  hipDeviceSynchronize();

  /*
  computeSurfaceTensionForce<<<blockSize, threadSize>>>(particlesDevice,
                                                         deltaTime);
  hipDeviceSynchronize();
  */

  computeGravityDevice<<<blockSize, threadSize>>>(particlesDevice, deltaTime);
  hipDeviceSynchronize();

  computePositionParallel<<<blockSize, threadSize>>>(particlesDevice,
                                                     deltaTime);
  hipDeviceSynchronize();

  computeWallConstraintDevice<<<blockSize, threadSize>>>(particlesDevice,
                                                         deltaTime);
  hipDeviceSynchronize();

  /* Copy Particle Positions to VBO positions array */
  hipGraphicsMapResources(1, &cudaVBOResource, 0);
  float *d_positions;
  size_t size;
  hipGraphicsResourceGetMappedPointer((void **)&d_positions, &size,
                                       cudaVBOResource);
  copyPositionToVBO<<<blockSize, threadSize>>>(d_positions, particlesDevice);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("cuda kernel error: %s\n", hipGetErrorString(err));
  }

  hipGraphicsUnmapResources(1, &cudaVBOResource, 0);
}

void SPHSimulator::updateScalarField() {

  const int threadSize = 8;
  dim3 dimBlock(threadSize, threadSize, threadSize);
  const int blockSize = (GRID_SIZE + threadSize - 1) / threadSize;
  dim3 dimGrid(blockSize, blockSize, blockSize);

  updateScalarFieldDevice<<<dimGrid, dimBlock>>>(
      colorFieldDevice, particlesDevice, GRID_SIZE, 1954.0);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("updateScalarFieldDevice error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}

VertexData SPHSimulator::extractSurface() {
  return marchingCubes->march(colorFieldDevice,
                              SPHSimulatorConstants::SURFACE_LEVEL);
}

void SPHSimulator::extractParticlePositions(unsigned int vbo) {

  // float *devicePtr = nullptr;
  // size_t numBytes;
  // hipGraphicsMapResources(1, &cudaVBOResource, 0);
  // hipGraphicsResourceGetMappedPointer((void **)&devicePtr, &numBytes,
  //                                      cudaVBOResource);

  // hipMemcpy(devicePtr, particlesDevice,
  //            sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES,
  //            hipMemcpyDeviceToDevice);

  // hipGraphicsUnmapResources(1, &cudaVBOResource, 0);
  // hipMemcpy(particles.data(), particlesDevice,
  //            sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES,
  //            hipMemcpyDeviceToHost);
  // vector<float> positions;
  // positions.reserve(SPHSimulatorConstants::NUM_PARTICLES * 3);
  // for (const auto &i : particles) {
  //   positions.push_back(i.position.x);
  //   positions.push_back(i.position.y);
  //   positions.push_back(i.position.z);
  // }
  // return positions;
}