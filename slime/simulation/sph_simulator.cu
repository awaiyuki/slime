#include "hip/hip_runtime.h"

#include "sph_simulator.cuh"
#include "sph_simulator_parallel.cuh"
#include <cstring>
#include <random>
#include <iostream>

using namespace slime;
using namespace std;

SPHSimulator::SPHSimulator() {
  random_device rd;
  mt19937 gen(rd());
  uniform_real_distribution<> dis(0.4f, 0.5f);

  for (int i = 0; i < SPHSimulatorConstants::NUM_PARTICLES; i++) {
    Particle particle;
    particle.id = i;

    float x = static_cast<float>(dis(gen));
    float y = static_cast<float>(dis(gen));
    float z = static_cast<float>(dis(gen));
    particle.position = make_float3(x, y, z);
    particle.velocity = make_float3(0, 0, 0);

    // cout << "initial position: " << x << y << z << endl;
    particle.mass = SPHSimulatorConstants::PARTICLE_MASS;
    particles.push_back(particle);
  }

  marchingCubes = make_unique<MarchingCubes>(GRID_SIZE);

  memset(colorField, 0, sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE);

  hipMalloc((void **)&particlesDevice,
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES);
  hipMalloc((void **)&colorFieldDevice,
             sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE);

  hipMemcpy(particlesDevice, particles.data(),
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES,
             hipMemcpyHostToDevice);
}

SPHSimulator::~SPHSimulator() {
  hipFree(particlesDevice);
  hipFree(colorFieldDevice);
}

void SPHSimulator::updateParticles(double deltaTime) {

  const int threadSize = 128;
  const int blockSize =
      (SPHSimulatorConstants::NUM_PARTICLES + threadSize - 1) / threadSize;
  computeDensityDevice<<<blockSize, threadSize>>>(particlesDevice);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("computeDensity error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();

  computePressureDevice<<<blockSize, threadSize>>>(particlesDevice);

  computePressureForceDevice<<<blockSize, threadSize>>>(particlesDevice,
                                                        deltaTime);
  hipDeviceSynchronize();

  computeViscosityForceDevice<<<blockSize, threadSize>>>(particlesDevice,
                                                         deltaTime);
  hipDeviceSynchronize();

  computeSurfaceTensionDevice<<<blockSize, threadSize>>>(particlesDevice,
                                                         deltaTime);
  hipDeviceSynchronize();

  /*
  computeSurfaceTensionForce<<<blockSize, threadSize>>>(particlesDevice,
                                                         deltaTime);
  hipDeviceSynchronize();
  */

  computeGravityDevice<<<blockSize, threadSize>>>(particlesDevice, deltaTime);
  hipDeviceSynchronize();

  computePositionParallel<<<blockSize, threadSize>>>(particlesDevice,
                                                     deltaTime);
  hipDeviceSynchronize();

  computeWallConstraintDevice<<<blockSize, threadSize>>>(particlesDevice,
                                                         deltaTime);
  hipDeviceSynchronize();
}

void SPHSimulator::updateScalarField() {

  const int threadSize = 8;
  dim3 dimBlock(threadSize, threadSize, threadSize);
  const int blockSize = (GRID_SIZE + threadSize - 1) / threadSize;
  dim3 dimGrid(blockSize, blockSize, blockSize);
  updateScalarFieldDevice<<<dimGrid, dimBlock>>>(colorFieldDevice,
                                                 particlesDevice, GRID_SIZE);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("updateScalarFieldDevice error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}

VertexData SPHSimulator::extractSurface() {
  return marchingCubes->march(colorFieldDevice,
                              SPHSimulatorConstants::SURFACE_LEVEL);
}

std::vector<float> SPHSimulator::extractParticlePositions() {

  hipMemcpy(particles.data(), particlesDevice,
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES,
             hipMemcpyDeviceToHost);
  vector<float> positions;
  positions.reserve(SPHSimulatorConstants::NUM_PARTICLES * 3);
  for (const auto &i : particles) {
    positions.push_back(i.position.x);
    positions.push_back(i.position.y);
    positions.push_back(i.position.z);
  }
  return positions;
}