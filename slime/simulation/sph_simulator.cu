#include "hip/hip_runtime.h"

#include "sph_simulator.cuh"
#include "sph_simulator_kernel.cuh"
#include <cstring>
#include <random>
#include <iostream>

using namespace slime;
using namespace std;

SPHSimulator::SPHSimulator() {
  random_device rd;
  mt19937 gen(rd());
  uniform_real_distribution<> dis(0.3f, 0.6f);

  for (int i = 0; i < SPHSimulatorConstants::NUM_PARTICLES; i++) {
    Particle particle;
    particle.id = i;

    float x = static_cast<float>(dis(gen));
    float y = static_cast<float>(dis(gen));
    float z = static_cast<float>(dis(gen));
    particle.position = glm::vec3(x, y, z);
    particle.velocity = glm::vec3(0, 0, 0);

    // cout << "initial position: " << x << y << z << endl;
    particle.mass = 1.0f;
    particles.push_back(particle);
  }

  memset(colorField, 0, sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE);

  hipMalloc((void **)&particlesDevice,
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES);
  hipMalloc((void **)&colorFieldDevice,
             sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE);

  hipMemcpy(particlesDevice, particles.data(),
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES,
             hipMemcpyHostToDevice);
}

SPHSimulator::~SPHSimulator() {
  hipFree(particlesDevice);
  hipFree(colorFieldDevice);
}

void SPHSimulator::updateParticles(double deltaTime) {
  updateParticlesDevice<<<1, SPHSimulatorConstants::NUM_PARTICLES>>>(
      particlesDevice, deltaTime);
}

void SPHSimulator::updateScalarField() {
  dim3 dimBlock(1, 1, 1);
  dim3 dimGrid(GRID_SIZE, GRID_SIZE, GRID_SIZE); // need to be updated
  updateScalarFieldDevice<<<dimBlock, dimGrid>>>(colorFieldDevice,
                                                 particlesDevice, GRID_SIZE);
  hipDeviceSynchronize();
  hipMemcpy(colorField, colorFieldDevice,
             sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE,
             hipMemcpyDeviceToHost);
}

float SPHSimulator::poly6Kernel(glm::vec3 r, float h) {
  float rMagnitude = glm::length(r);
  if (rMagnitude > h)
    return 0.0f;

  return 315.0f / (64.0f * PI * glm::pow(h, 9)) *
         glm::pow(h * h - rMagnitude * rMagnitude, 3);
}

float SPHSimulator::spikyKernel(glm::vec3 r, float h) { return 0.0f; }

float SPHSimulator::gradientSpikyKernel(glm::vec3 r, float h) {
  float rMagnitude = glm::length(r);
  if (rMagnitude > h)
    return 0.0f;

  return -45.0f / (PI * glm::pow(h, 6)) * glm::pow(h - rMagnitude, 2);
}

float SPHSimulator::viscosityKernel(glm::vec3 r, float h) { return 0.0f; }

float SPHSimulator::laplacianViscosityKernel(glm::vec3 r, float h) {
  float rMagnitude = glm::length(r);
  if (rMagnitude > h)
    return 0.0f;

  return 45 / (PI * glm::pow(h, 6)) * (h - rMagnitude);
}

void SPHSimulator::computeDensity() {
  for (auto &i : particles) {
    i.density = 0.0f;
    for (auto &j : particles) {
      if (i == j)
        continue;

      auto r = j.position - i.position;
      i.density +=
          j.mass * poly6Kernel(r, SPHSimulatorConstants::SMOOTHING_RADIUS);
    }
  }
}

void SPHSimulator::computePressureForce(double deltaTime) {
  for (auto &i : particles) {
    i.pressure = SPHSimulatorConstants::GAS_CONSTANT *
                 (i.density - SPHSimulatorConstants::REST_DENSITY);
  }

  for (auto &i : particles) {
    glm::vec3 pressureForce = glm::vec3(0.0f, 0.0f, 0.0f);
    for (auto &j : particles) {
      if (i == j)
        continue;

      if (j.density < EPSILON)
        continue;

      auto r = j.position - i.position;
      pressureForce +=
          -glm::normalize(r) * j.mass * (i.pressure + j.pressure) /
          (2.0f * j.density) *
          gradientSpikyKernel(r, SPHSimulatorConstants::SMOOTHING_RADIUS);
    }
    auto acceleration = pressureForce / i.mass;
    auto deltaVelocity = acceleration * float(deltaTime);
    i.velocity += deltaVelocity;
  }
}

void SPHSimulator::computeViscosityForce(double deltaTime) {
  for (auto &i : particles) {
    glm::vec3 viscosityForce = glm::vec3(0.0f, 0.0f, 0.0f);
    for (auto &j : particles) {
      if (i == j)
        continue;

      if (j.density < EPSILON)
        continue;

      auto r = j.position - i.position;
      viscosityForce +=
          j.mass * (j.velocity - i.velocity) / j.density *
          laplacianViscosityKernel(r, SPHSimulatorConstants::SMOOTHING_RADIUS);
    }
    viscosityForce *= SPHSimulatorConstants::VISCOSITY_COEFFICIENT;

    auto acceleration = viscosityForce / i.mass;
    auto deltaVelocity = acceleration * float(deltaTime);
    i.velocity += deltaVelocity;
  }
}

void SPHSimulator::computeGravity(double deltaTime) {
  for (auto &i : particles) {
    auto acceleration = glm::vec3(0, -0.098f, 0);
    auto deltaVelocity = acceleration * float(deltaTime);
    i.velocity += deltaVelocity;
  }
}

void SPHSimulator::computeWallConstraint(double deltaTime) {

  /* Spring-Damper Collision */

  for (auto &i : particles) {
    const float FLOOR_CONSTRAINT = -3.0f;
    const float CEILING_CONSTRAINT = 3.0f;
    const float SPRING_CONSTANT = 500.0f;
    const float DAMPING = 1.0f;
    if (i.position.x < FLOOR_CONSTRAINT) {
      auto deltaVelocity =
          (SPRING_CONSTANT * (FLOOR_CONSTRAINT - i.position.x) +
           DAMPING * i.velocity.x) *
          float(deltaTime);
      i.velocity.x += deltaVelocity;
    }

    if (i.position.x > CEILING_CONSTRAINT) {
      auto deltaVelocity =
          (SPRING_CONSTANT * (i.position.x - CEILING_CONSTRAINT) +
           DAMPING * i.velocity.x) *
          float(deltaTime);
      i.velocity.x -= deltaVelocity;
    }
    if (i.position.y < FLOOR_CONSTRAINT) {
      auto deltaVelocity =
          (SPRING_CONSTANT * (FLOOR_CONSTRAINT - i.position.y) +
           DAMPING * i.velocity.y) *
          float(deltaTime);
      i.velocity.y += deltaVelocity;
    }

    if (i.position.y > CEILING_CONSTRAINT) {
      auto deltaVelocity =
          (SPRING_CONSTANT * (i.position.y - CEILING_CONSTRAINT) +
           DAMPING * i.velocity.y) *
          float(deltaTime);
      i.velocity.y -= deltaVelocity;
    }
    if (i.position.z < FLOOR_CONSTRAINT) {
      auto deltaVelocity =
          (SPRING_CONSTANT * (FLOOR_CONSTRAINT - i.position.z) +
           DAMPING * i.velocity.z) *
          float(deltaTime);
      i.velocity.z += deltaVelocity;
    }

    if (i.position.z > CEILING_CONSTRAINT) {
      auto deltaVelocity =
          (SPRING_CONSTANT * (i.position.z - CEILING_CONSTRAINT) +
           DAMPING * i.velocity.z) *
          float(deltaTime);
      i.velocity.z -= deltaVelocity;
    }
  }
}
std::vector<MarchingCubes::Triangle> SPHSimulator::extractSurface() {
  MarchingCubes marchingCubes;
  return marchingCubes.march(colorField, SPHSimulatorConstants::SURFACE_LEVEL);
}

std::vector<glm::vec3> SPHSimulator::extractParticlePositions() {
  vector<glm::vec3> positions;
  for (auto &i : particles) {
    positions.push_back(i.position);
  }
  return positions;
}