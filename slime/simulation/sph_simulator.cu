#include "hip/hip_runtime.h"

#include "sph_simulator.cuh"
#include "sph_simulator_device.cuh"
#include <cstring>
#include <random>
#include <iostream>
#include <stdio.h>
#include <glad/gl.h>
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <slime/utility/cuda_debug.cuh>

using namespace slime;
using namespace slime::SPHSimulatorConstants;
using namespace std;

SPHSimulator::SPHSimulator(const unsigned int vbo)
    : hashKeys(SPHSimulatorConstants::NUM_PARTICLES, 0),
      hashIndices(SPHSimulatorConstants::NUM_PARTICLES, 0),
      bucketStart(SPHSimulatorConstants::NUM_PARTICLES, 0),
      bucketEnd(SPHSimulatorConstants::NUM_PARTICLES, 0) {
  random_device rd;
  mt19937 gen(rd());
  uniform_real_distribution<> dis(-0.1f, 0.1f);
  particles.reserve(SPHSimulatorConstants::NUM_PARTICLES);
  for (int i = 0; i < SPHSimulatorConstants::NUM_PARTICLES; i++) {
    Particle particle;
    particle.id = i;

    float x = static_cast<float>(dis(gen));
    float y = static_cast<float>(dis(gen));
    float z = static_cast<float>(dis(gen));
    particle.position = make_float3(x, y, z);
    particle.velocity = make_float3(0, 0, 0);

    // cout << "initial position: " << x << y << z << endl;
    particle.mass = SPHSimulatorConstants::PARTICLE_MASS;
    particles.push_back(particle);
  }

  marchingCubes = make_unique<MarchingCubes>(GRID_SIZE);

  std::vector<float> scalarField(GRID_SIZE * GRID_SIZE * GRID_SIZE, 0);

  hipMalloc((void **)&d_particles,
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES);
  hipMalloc((void **)&d_scalarField,
             sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE);

  hipMemcpy(d_particles, particles.data(),
             sizeof(Particle) * SPHSimulatorConstants::NUM_PARTICLES,
             hipMemcpyHostToDevice);

  hipMemcpy(d_scalarField, scalarField.data(),
             sizeof(float) * GRID_SIZE * GRID_SIZE * GRID_SIZE,
             hipMemcpyHostToDevice);
  hipGraphicsGLRegisterBuffer(&cudaVBOResource, vbo, cudaGraphicsMapFlagsNone);

  raw_hashKeys = thrust::raw_pointer_cast(hashKeys.data());
  raw_hashIndices = thrust::raw_pointer_cast(hashIndices.data());
  raw_bucketStart = thrust::raw_pointer_cast(bucketStart.data());
  raw_bucketEnd = thrust::raw_pointer_cast(bucketEnd.data());
}

SPHSimulator::~SPHSimulator() {
  hipFree(d_particles);
  hipFree(d_scalarField);
}

std::vector<Particle> *SPHSimulator::getParticlesPointer() {
  return &particles;
}

void SPHSimulator::updateParticles(double deltaTime) {

  const int threadSize = 128;
  const int blockSize =
      (SPHSimulatorConstants::NUM_PARTICLES + threadSize - 1) / threadSize;

  /* Updating Spatial Hashing */

  cout << "check1" << endl;
  updateSpatialHashDevice<<<blockSize, threadSize>>>(d_particles, raw_hashKeys,
                                                     raw_hashIndices);
  hipDeviceSynchronize();

  printCudaError("updateSpatialHash or before");

  cout << "check2" << endl;
  thrust::sort_by_key(hashKeys.begin(), hashKeys.end(), hashIndices.begin());
  hipDeviceSynchronize();

  printCudaError("sortbykey");
  cout << "check3" << endl;

  raw_hashKeys = thrust::raw_pointer_cast(hashKeys.data());
  raw_hashIndices = thrust::raw_pointer_cast(hashIndices.data());

  updateHashBucketDevice<<<blockSize, threadSize>>>(
      raw_hashKeys, raw_hashIndices, raw_bucketStart, raw_bucketEnd);
  hipDeviceSynchronize();
  printCudaError("updateHashBucket");

  /* Updating Particle attributes */

  computeDensityDevice<<<blockSize, threadSize>>>(
      d_particles, raw_hashIndices, raw_bucketStart, raw_bucketEnd);
  hipDeviceSynchronize();
  printCudaError("computeDensity");

  computePressureDevice<<<blockSize, threadSize>>>(d_particles);

  computePressureForceDevice<<<blockSize, threadSize>>>(
      d_particles, raw_hashIndices, raw_bucketStart, raw_bucketEnd, deltaTime);
  hipDeviceSynchronize();

  printCudaError("computePressureForce");
  computeViscosityForceDevice<<<blockSize, threadSize>>>(
      d_particles, raw_hashIndices, raw_bucketStart, raw_bucketEnd, deltaTime);
  hipDeviceSynchronize();
  printCudaError("computeViscosityForce");

  computeSurfaceTensionDevice<<<blockSize, threadSize>>>(
      d_particles, raw_hashIndices, raw_bucketStart, raw_bucketEnd, deltaTime);
  hipDeviceSynchronize();
  printCudaError("computeSurfaceTensionForce");

  computeGravityDevice<<<blockSize, threadSize>>>(d_particles, deltaTime);
  hipDeviceSynchronize();

  computePositionDevice<<<blockSize, threadSize>>>(d_particles, deltaTime);
  hipDeviceSynchronize();

  computeWallConstraintDevice<<<blockSize, threadSize>>>(d_particles,
                                                         deltaTime);
  hipDeviceSynchronize();

  /* Copying Particle Positions to VBO positions array */

  cout << "check4" << endl;
  hipGraphicsMapResources(1, &cudaVBOResource, 0);
  float *d_positions;
  size_t size;
  hipGraphicsResourceGetMappedPointer((void **)&d_positions, &size,
                                       cudaVBOResource);
  copyPositionToVBODevice<<<blockSize, threadSize>>>(d_positions, d_particles);
  hipDeviceSynchronize();

  hipGraphicsUnmapResources(1, &cudaVBOResource, 0);
}

void SPHSimulator::updateScalarField() {
  /* Need to debug */

  const int threadSize = 8;
  dim3 dimBlock(threadSize, threadSize, threadSize);
  const int blockSize = (GRID_SIZE + threadSize - 1) / threadSize;
  dim3 dimGrid(blockSize, blockSize, blockSize);

  updateScalarFieldDevice<<<dimGrid, dimBlock>>>(
      d_scalarField, d_particles, GRID_SIZE,
      1954.0); // need to investigate normalization methods.

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("updated_scalarField error: %s\n", hipGetErrorString(err));
  }
  hipDeviceSynchronize();
}

VertexData SPHSimulator::extractSurface() {
  return marchingCubes->march(d_scalarField,
                              SPHSimulatorConstants::SURFACE_LEVEL);
}
